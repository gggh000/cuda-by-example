#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#define N 1000

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void add(int *a, int *b, int *c)
{
	int tid = threadIdx.x;								// current thread's x dimension.

	if (tid < N)
		c[tid] = a[tid] + b[tid];						// add as long as it is smaller than input vector.,	
}

int main()
{
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	int stat;

	// Start allocating memory for 3 vectors in GPU.

	stat = hipMalloc((void**)&dev_a, N * sizeof(int));
	stat = hipMalloc((void**)&dev_b, N * sizeof(int));
	stat = hipMalloc((void**)&dev_c, N * sizeof(int));

	// Construct vectors values for a and b vector.

	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i*i;
	}

	// Copy the summing vectors to device. 

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	add << <1, N>> > (dev_a, dev_b, dev_c);

	// Copy the summed vector back to host.

	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	// Print the vector now.

	for (int i = 0; i < N; i++)
		printf("\n%d + %d = %d", a[i], b[i], c[i]);

	// Release device memory. 
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	getchar();
	return 0;
}
