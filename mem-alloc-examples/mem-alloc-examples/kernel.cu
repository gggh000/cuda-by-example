
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void kerneladd(int  *dev_c)
{
    *dev_c = 1000;
//    *pMemAddr = 1001;
}

int main()
{
	int * dev_c;
	int a = 300;
	int memAddr = 0;
	int stat;
	
	printf("Size of int: %d.\n", sizeof(int));
	printf("a before kernel call: %u.\n", a);
	printf("1. hipMalloc example, default parameters.\n");
	//printf("dev_c host address before cudaMalloc: 0x%08x\n", dev_c);

	hipMalloc((void**)&dev_c, sizeof(int));

	//printf("dev_c host address after cudaMalloc: 0x%08x\n", dev_c);

	hipMemcpy(dev_c, &a, sizeof(int), hipMemcpyHostToDevice);
	kerneladd <<<1, 1>>>  (dev_c);
	hipMemcpy(&a, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("a after kernel call: %u.\n", a);
	//printf("memAddr %08x", memAddr);

	hipFree(dev_c);
	getchar();
	return 0;
}
