
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void kernel(int * a, int * pMemAddr)
{
    *a = 1000;
	*pMemAddr = 1001;
}

int main()
{
	int * dev_a  = 0;
	int a = 100;
	int memAddr = 0;

	printf("Size of int: %d", sizeof(int));
	
	printf("a before kernel call: %u.\n", a);
	printf("1. hipMalloc example, default parameters.\n");
	printf("dev_a host address before: 0x%08x\n", dev_a);
	hipMalloc((void**)&dev_a, sizeof(int));
	printf("dev_a host address after: 0x%08x\n", dev_a);

	hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);
	kernel <<<1, 1>>>  (dev_a, &memAddr);
	hipMemcpy(&a, dev_a, sizeof(int), hipMemcpyDeviceToHost);

	printf("a after kernel call: %u.\n", a);
	printf("memAddr %08x", memAddr);

	hipFree(dev_a);
	getchar();
	return 0;
}
