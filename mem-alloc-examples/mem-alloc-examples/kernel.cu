
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	int * dev_a  = 0;

	printf("1. hipMalloc example, default parameters.\n");
	printf("dev_a before: 0x%08x\n", dev_a);
	hipMalloc((void**)&dev_a, 1024);
	printf("dev_a after: 0x%08x\n", dev_a);
	hipFree(dev_a);
	getchar();
	return 0;
}
