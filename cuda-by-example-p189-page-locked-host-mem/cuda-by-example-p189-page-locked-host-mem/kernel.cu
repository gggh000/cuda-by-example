
#include "hip/hip_runtime.h"


#include <stdio.h>
#define SIZE (100*1024*1024)


float cuda_malloc_test(int size, bool up, bool hostAlloc = false) {
	hipEvent_t start, stop;
	int *a, *dev_a;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (hostAlloc) {
		a = (int*)hipHostAlloc((void**)&a, size * sizeof(*a), hipHostMallocDefault);
	} else {
		a = (int*)malloc(size * sizeof(*a));
	}
	hipMalloc((void**)&dev_a, size * sizeof(*dev_a));

	hipEventRecord(start, 0);
	for (int i = 0; i < 100; i++) {
		if (up)
			hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice);
		else
			hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	if (hostAlloc) {
		hipHostFree(dev_a);
	} else {
		hipFree(dev_a);
	}
	hipFree(a);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return elapsedTime;		
}

int main()
{
	float elapsedTime;
	
	printf("hipMalloc test:\n");
	float MB = (float)100 * SIZE * sizeof(int) / 1024 / 1024;
	elapsedTime = cuda_malloc_test(SIZE, true);
	printf("Time using cudaMalloc(up): %3.1f ms.\n", elapsedTime);
	printf("MB/s during copy up: %3.1f.\n", MB / (elapsedTime / 1000));
	elapsedTime = cuda_malloc_test(SIZE, false);
	printf("Time using cudaMalloc(down): %3.1f ms.\n", elapsedTime);
	printf("MB/s during copy up: %3.1f.\n", MB / (elapsedTime / 1000));

	printf("cudaHostalloc test:\n");
	elapsedTime = cuda_malloc_test(SIZE, true, 1);
	printf("Time using cudaMalloc(up): %3.1f ms.\n", elapsedTime);
	printf("MB/s during copy up: %3.1f.\n", MB / (elapsedTime / 1000));
	elapsedTime = cuda_malloc_test(SIZE, false, 1);
	printf("Time using cudaMalloc(down): %3.1f ms.\n", elapsedTime);
	printf("MB/s during copy up: %3.1f.\n", MB / (elapsedTime / 1000));
}
