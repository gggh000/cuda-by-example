#include "common/book.h"



int main( void ) {
	hipDeviceProp_t prop;

	int count;
	HANDLE_ERROR( hipGetDeviceCount( &count ) );

	for( int i = 0; i < count; i++ ) {
		HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
		printf( "Name: %s\n", prop.name);
	}
}
