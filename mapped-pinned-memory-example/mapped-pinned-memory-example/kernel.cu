#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#define N 10

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void add(int *a, int *b, int *c)
{
	int tid = threadIdx.x;								// current thread's x dimension.

	if (tid < N)
		c[tid] = a[tid] + b[tid];						// add as long as it is smaller than input vector.,	
}

int main()
{
	//int a[N], b[N], c[N];
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;
	int stat[2];

	dev_a = NULL;
	dev_b = NULL;
	dev_c = NULL;

	// Start allocating memory for 3 vectors in GPU.

	stat[0] = hipHostAlloc((void**)&a, N * sizeof(int), hipHostMallocMapped);
	stat[1] = hipHostAlloc((void**)&b, N * sizeof(int), hipHostMallocMapped);
	stat[2] = hipHostAlloc((void**)&c, N * sizeof(int), hipHostMallocMapped);

	printf("\n1. stat: %d", stat[0]);
	printf("\n1. stat: %d", stat[1]);
	printf("\n1. stat: %d", stat[2]);

	//checkCudaErrors(hipHostGetDevicePointer((void **)&dev_a, (void*)a, 0));
	//checkCudaErrors(hipHostGetDevicePointer((void **)&dev_b, (void*)b, 0));
	//checkCudaErrors(hipHostGetDevicePointer((void **)&dev_c, (void*)c, 0));
	
	printf("\ndev_a:c: 0x%08x, 0x%08x, 0x%08x", dev_a, dev_b, dev_c);
	stat[0] = hipHostGetDevicePointer((void **)&dev_a, (void*)a, 0);
	stat[1] = hipHostGetDevicePointer((void **)&dev_b, (void*)b, 0);
	stat[2] = hipHostGetDevicePointer((void **)&dev_c, (void*)c, 0);
	printf("\ndev_a:c: 0x%08x, 0x%08x, 0x%08x", dev_a, dev_b, dev_c);

	printf("\n2. stat: %d", stat[0]);
	printf("\n2. stat: %d", stat[1]);
	printf("\n2. stat: %d", stat[2]);

	printf("\n0x%08x", a);
	printf("\n0x%08x", dev_a);

	// Construct vectors values for a and b vector.
	
	for (int i = 0; i < N; i++) {
		dev_a[i] = i;
		dev_b[i] = i*i;
	}
	
	// Copy the summing vectors to device. 

	//hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	add << <1, N >> > (dev_a, dev_b, dev_c);

	// Copy the summed vector back to host.

	//hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	// Print the vector now.

	for (int i = 0; i < N; i++)
		printf("\n%d + %d = %d", dev_a[i], dev_b[i], dev_c[i]);

	// Release device memory. 
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	getchar();
	return 0;
}
