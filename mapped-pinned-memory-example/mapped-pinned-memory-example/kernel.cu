
#include "hip/hip_runtime.h"


#include <stdio.h>
#define N 10
#define CONFIG_ENABLE_PINNED_MAPPED_MEMO 0

__global__ void add(int *a, int *b, int *c)
{
	int tid = threadIdx.x;								// current thread's x dimension.
	printf("\nadd(): tid: %d. 0x%08x, 0x%08x, 0x%08x %d, %d, %d.", tid, a, b, c, *a, *b, *c);
	if (tid < N) {
		a[tid] += 100;
		c[tid] = a[tid] + b[tid];						// add as long as it is smaller than input vector.,	
		//c[tid] = (int)&c[tid];
	}
}

int main()
{
#if CONFIG_ENABLE_PINNED_MAPPED_MEMO == 1
	int *a, *b, *c;
#else
	int a[N], b[N], c[N];
#endif

	int *dev_a, *dev_b, *dev_c;
	int stat[2];

	dev_a = NULL;
	dev_b = NULL;
	dev_c = NULL;

	// Start allocating memory for 3 vectors in GPU.

	if (CONFIG_ENABLE_PINNED_MAPPED_MEMO == 1) {
		printf("CONFIG_ENABLE_PINNED_MAPPED_MEMO on...");
		stat[0] = hipHostAlloc((void**)&a, N * sizeof(int), hipHostMallocDefault);
		stat[1] = hipHostAlloc((void**)&b, N * sizeof(int), hipHostMallocDefault);
		stat[2] = hipHostAlloc((void**)&c, N * sizeof(int), hipHostMallocDefault);
	}	else {
		printf("CONFIG_ENABLE_PINNED_MAPPED_MEMO off...");
		stat[0] = hipMalloc((void**)&dev_a, N * sizeof(int));
		stat[1] = hipMalloc((void**)&dev_b, N * sizeof(int));
		stat[2] = hipMalloc((void**)&dev_c, N * sizeof(int));
	}

	for (int i = 0; i < 3; i++)
		printf("\n%d. stat: %d", i, stat[i]);

	if (CONFIG_ENABLE_PINNED_MAPPED_MEMO == 1) {
		printf("\ndev_a:c: 0x%08x, 0x%08x, 0x%08x", dev_a, dev_b, dev_c);
		stat[0] = hipHostGetDevicePointer((void **)&dev_a, (void*)a, 0);
		stat[1] = hipHostGetDevicePointer((void **)&dev_b, (void*)b, 0);
		stat[2] = hipHostGetDevicePointer((void **)&dev_c, (void*)c, 0);
		printf("\ndev_a:c: 0x%08x, 0x%08x, 0x%08x", dev_a, dev_b, dev_c);

		for (int i = 0; i < 3; i++)
			printf("\n%d. stat: %d", i, stat[i]);
		printf("\n0x%08x", a);
		printf("\n0x%08x", dev_a);
	}

	// Construct vectors values for a and b vector.
	
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i*i;
		c[i] = -1;
	}
	
	// Copy the summing vectors to device. 

	if (CONFIG_ENABLE_PINNED_MAPPED_MEMO == 0) {
		hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	}
	printf("\nResult before:");
	for (int i = 0; i < N; i++) {
		if (CONFIG_ENABLE_PINNED_MAPPED_MEMO == 1) {
			printf("\n%d + %d = %d", dev_a[i], dev_b[i], dev_c[i]);
		}
		else {
			printf("\n%d + %d = %d", a[i], b[i], c[i]);
		}
	}

	add <<<1, N >>> (dev_a, dev_b, dev_c);

	// Copy the summed vector back to host.

	// Print the vector now.

	if (CONFIG_ENABLE_PINNED_MAPPED_MEMO != 1) {
		hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	}

	printf("\nResult before:");
	for (int i = 0; i < N; i++) {
		if (CONFIG_ENABLE_PINNED_MAPPED_MEMO == 1) {
			printf("\n%d + %d = %d", dev_a[i], dev_b[i], dev_c[i]);
		} else {
			printf("\n%d + %d = %d", a[i], b[i], c[i]);
		}
	}

	// Release device memory. 

	if (CONFIG_ENABLE_PINNED_MAPPED_MEMO == 1) {
		hipHostFree(dev_a);
		hipHostFree(dev_b);
		hipHostFree(dev_c);
	} else {
		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);
	}

	getchar();
	return 0;
}
