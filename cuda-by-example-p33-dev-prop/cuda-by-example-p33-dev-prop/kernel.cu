#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <string.h>


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	hipDeviceProp_t prop;
	int dev;
	int stat; 

	hipGetDevice(&dev);
	printf("ID of current CUDA device: %d\n", dev);

	// causes build error. 
	memset ( &prop, 0, sizeof(hipDeviceProp_t));

	prop.major = 1;
	prop.minor = 3;
	hipChooseDevice(&dev, &prop);
	printf("ID of current CUDA device closest to revision: %d\n", dev);

	printf("\nComputeMode: %d", prop.computeMode);
	printf("\ncanMapHostMemory: %d", prop.canMapHostMemory);
	printf("\nBus Device Domain: %d-%d-%d", prop.pciBusID, prop.pciDeviceID, prop.pciDomainID);

	getchar();
 return 0;
}
